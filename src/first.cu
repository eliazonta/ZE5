#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

__global__ void kernel(void){
    printf("hello ! [from device/ thread]\n");
}

__global__ void gpu_print(void){
    printf("Hello from thread [%d, %d]\n", threadIdx.x, blockIdx.x);
}
int main(void){
	kernel<<<1, 1>>>();
	printf("Hello World! [from host]\n");

    gpu_print<<<2, 1>>>();
    cudaDeviceSyncronize();
	return 0;
}
