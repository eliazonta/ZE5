#include "hip/hip_runtime.h"
#include "../include/dep.h"

__global__ void kernel(void){
    printf("Hello from the kernel! \n");
}

__global__ void gpu_print(void){
    printf("Hello from thread [%d, %d]\n", threadIdx.x, blockIdx.x);
}

int main(void){
    dtype a = 10;
    printf("Hello [from host], a w/ dtype  = %d\n", a);

    kernel<<<1, 1>>>();
    hipDeviceSynchronize();

    gpu_print<<<2, 1>>>();
    hipDeviceSynchronize();
}