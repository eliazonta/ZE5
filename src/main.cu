#include "hip/hip_runtime.h"
#include "../include/dep.h"

__global__ void gpu_print(void){
    printf("Hello from thread [%d, %d]\n", threadIdx.x, blockIdx.x);
}

int main(int argc, char** argv)
{
    GpuTime time();
    // parse data from the data file -> create a parser
    // alg
    // external check
    // benckmarking (how)

//    printf("Hello [from host], a w/ dtype  = %d\n", a);


    gpu_print<<<2, 1>>>();
    hipDeviceSynchronize();
    return 0;
}