#include "hip/hip_runtime.h"
#include "../include/dep.h"

__global__ 
void gpu_print(void){
    printf("Hello from thread [%d, %d]\n", threadIdx.x, blockIdx.x);
}
__global__ 
void VecAdd(float* a, float* b, float* c, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < N){
        c[i] = a[i] + b[i];
    }
}

int main(int argc, char** argv)
{
    GpuTime time();
    // parse data from the data file -> create a parser
    // CSR ? COO ? what's better ? 
    // alg {pain}
    // external check {py / networkx}
    // benckmarking [gpu timer content is sufficient ?]

    int N = 5;
    size_t size = sizeof(float) * N;

    float* a = (float*)malloc(size);
    float* b = (float*)malloc(size);
    float* c = (float*)malloc(size);

    for(int i = 0; i < N; ++i){
        *(a + i) = (float)i;
        *(b + i) = (float)i * 2;
    }

    float* d_a;hipMalloc(&d_a, size);
    float* d_b;hipMalloc(&d_b, size);
    float* d_c;hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1)/ threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("A : ");
    for (int i = 0; i < N; ++i)
    {
        printf("%f - ", *(a + i));
    }
    printf("\n");
    printf("B : ");
    for (int i = 0; i < N; ++i)
    {
        printf("%f - ", *(b + i));
    }
    printf("\n");
    
    printf("C : ");
    for (int i = 0; i < N; ++i)
    {
        printf("%f - ", *(c + i));
    }
    printf("\n");
    
   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);
    free(a);free(b);free(c);

    return 0;
}